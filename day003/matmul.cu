
#include <hip/hip_runtime.h>
__global__
void matrixMulKernel(float* M, float* N, float* P, int Width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // row and col thread indices are also the indices for P matrix

    // for a 4x4 matrix P
    // Width = 4
    // BLOCK_SIZE = 2

    if ((row < Width) && (col < Width)) {
        float Pvalue = 0;
        for (int k = 0; k < Width; k++) {
            Pvalue += M[row * Width + k] * N[k * Width + col];
        }
        P[row * Width + col] = Pvalue;
    }
}